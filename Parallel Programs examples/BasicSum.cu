#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdio.h>


// global function is the main of the gpu
__global__ void suma(int *a, int *b, int *c){
	*c = *a + *b;
}


int main(){
	//variables for the cpu (host)
	int a, b, c;
	
	//variables for the gpu (device)
	int *d_a, *d_b, *d_c;
	
	int size = sizeof(int);
	
	//reserve memory in the GPU
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	
	//test values
	a = 10;
	b = 20;

	//copy the values from the cpu(host) to the gpu(device)
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
	//kernel call with 1 thread and 1 block
	suma << <1, 1 >> >(d_a, d_b, d_c);

	//copy the values from the gpu(device) to the cpu(host)
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	
	printf("result is  %d\n", c);

	//free memory from the cuda pointers
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}
