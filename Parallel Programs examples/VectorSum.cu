#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N (4096*4096)
#define HILOS_POR_BLOQUE 512

__global__ void sumaenlagpu(int *a, int *b, int *c, int n){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index < n){
		c[index] = a[index] + b[index];
	}
}

void sumaenlacpu(int *a, int *b, int *c, int n){
	for(int i = 0; i< n; i++){
		c[i] = a[i] + b[i];
	}
}

void numerosAleatorios(int *a, int n){
	for (int i = 0; i < n; i++){
		a[i] = rand() % 100000;
	}
}

int comparaeneteros(int *a, int *b, int n){
	int pass = 0;
	for (int i = 0; i < n; i++){
		if (a[i] != b[i])
			printf("Los valores  en a[%i] = %i y en b[%i] = %i", i, a[i], i, b[i]);
	}
	if (pass == 0){
		printf("Comprobacion aceptada \n");
	}
	else{
		printf("No paso la comprobacion \n");
	}
	return pass;
}

int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int tam = N*sizeof(int);
	
	
	hipMalloc((void**)&d_a, tam);
	hipMalloc((void**)&d_b, tam);
	hipMalloc((void**)&d_c, tam);

	a = (int*)malloc(tam);
	b = (int*)malloc(tam);
	c = (int*)malloc(tam);

	numerosAleatorios(a,N);
	numerosAleatorios(b,N);
	
	hipMemcpy(d_a, a, tam, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, tam, hipMemcpyHostToDevice);

	clock_t tiempogpu = clock();

	
	sumaenlagpu << <N / HILOS_POR_BLOQUE, HILOS_POR_BLOQUE >> >(d_a, d_b, d_c, N);
	printf("Tiempo transcurrido al procesador en GPU: %f\n", ((double)clock() - tiempogpu) / CLOCKS_PER_SEC);
	
	hipMemcpy(c, d_c, tam, hipMemcpyDeviceToHost);
	
	int *c_h;
	c_h = (int *)malloc(tam);
	
	clock_t tiempocpu = clock();
	sumaenlacpu(a, b, c_h, N);
	printf("Tiempo transcurrido al procesador en CPU: %f\n", ((double)clock() - tiempocpu) / CLOCKS_PER_SEC);
	comparaeneteros(c, c_h, N);

	
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
