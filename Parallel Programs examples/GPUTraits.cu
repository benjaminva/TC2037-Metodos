#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdio.h>


int main(){

	hipDeviceProp_t prop;  // Object that has access to the gpus traits
	
	int count; 
	hipGetDeviceCount(&count);
	
	for (int i = 0; i < count; i++){
		hipGetDeviceProperties(&prop, i);
		printf("name %s \n", prop.name);
		printf("clockrate %i \n", prop.clockRate);
		printf("maxGRID Size %i %i %i\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("max Thread Dim %i %i %i \n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("max Threads per Block Dim  %i \n", prop.maxThreadsPerBlock);
		printf("total global memory bytes %lli \n", (long long int)prop.totalGlobalMem);
		printf("total global memory MB %lli \n", ((long long int)prop.totalGlobalMem)/(1024*1024));
		printf("total global memory GB %lli \n", ((long long int)prop.totalGlobalMem)/(1024*1024*1024));
		printf("wrap size %i \n", prop.warpSize);
	
	}
}
