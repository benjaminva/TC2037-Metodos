#include "hip/hip_runtime.h"
//Matrix linearization

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N 4096  //size of problem will be N*N matrix
#define THREADS_PER_BLOCK 512   //size of block

__global__ void add(int *a, int *b, int *c, int max){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int id = index;
	while (id < max){
		c[id] = a[id] + b[id];
		id =  id + blockDim.x* gridDim.x;    
	}
}

void fill_mat(int *mat ){
	int c = 0;
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			mat[j + i*N] = 1;
		}
	}
}


void print_mat(int *mat){
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++){
			printf("%i \t", mat[j+i*N]);
		}
		printf("\n");
	}
	printf("\n");
}

int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int tam = N*N*sizeof(int);

	hipMalloc((void**)&d_a, tam);
	hipMalloc((void**)&d_b, tam);
	hipMalloc((void**)&d_c, tam);


	a = (int*)malloc(tam);
	b = (int*)malloc(tam);
	c = (int*)malloc(tam);

	fill_mat(a);
	fill_mat(b);
	
	//print_mat(a);
	//print_mat(b);

	hipMemcpy(d_a, a, tam, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, tam, hipMemcpyHostToDevice);


	add << <N*N / THREADS_PER_BLOCK,  THREADS_PER_BLOCK >> >(d_a, d_b, d_c, N*N);

	hipMemcpy(c, d_c, tam, hipMemcpyDeviceToHost);
	
	//print_mat(c);
	

	long int i, acum = 0;

	for(i = 0; i< N*N; i++){
		acum = acum + c[i];
	}

	printf("\n addition of values %i \n", acum);

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
